//nvcc testcuFFT.cu -o testcuFFT -lcufft
#include <iostream>
#include <vector>
#include <complex>
#include <cmath> // for sin()
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

int main()
{
    std::cout << "------------------test complex----------------------" << std::endl;
  {
    const int                         Nx    = 5;
    const int                         Ny    = 5;
    const int                         Nz    = 5;
    double                            Lx    = 1.0;
    const int                         procs = 1;
    const int                         rank  = 0;
    const double                      dx = 1.0, dy = 1.0, dz = 1.0;
    std::vector<std::complex<double>> data_cpu_in(Nx * Ny * (Nz + 2));

    for (int64_t k = 0; k < Nz; k++) {
      for (int64_t j = 0; j < Ny; j++) {
        for (int64_t i = 0, l = ((Nx / procs) * rank); i < (Nx / procs), l < ((Nx / procs) * (rank + 1)); i++, l++) {
          //ys           int idx = x + Nx * (y + Ny * z);
          double freq = 1.0; /*  */

          double dx     = 1.0 / Nx; // 1 wave divided by number of points
          double x_phys = i * dx;
          double kn     = 2.0 * M_PI * freq;

          reinterpret_cast<double*>(data_cpu_in.data())[(k * Ny * (Nz + 2)) + (j * (Nz + 2)) + i] =
              //ys 8 * ((sin(1.0 * l * dx) * sin(2.0 * j * dy) * sin(3.0 * k * dz)) +
              //ys      (sin(4.0 * l * dx) * sin(5.0 * j * dy) * sin(6.0 * k * dz)));
              8 * sin(kn * x_phys);
        }
      }
    }

    for (int64_t k = 0; k < Nz; k++) {
      for (int64_t j = 0; j < Ny; j++) {
        for (int64_t i = 0, l = ((Nx / procs) * rank); i < (Nx / procs), l < ((Nx / procs) * (rank + 1)); i++, l++) {
          auto& value = data_cpu_in[(k * Ny * (Nz + 2)) + (j * (Nz + 2)) + i];
          std::cout << "data_cpu_in[" << i << "," << j << "," << k << "] = "
                    << "Real: " << value.real() << ", Imag: " << value.imag() << std::endl;
        }
      }
    }

    std::cout << "----------------------------------------" << std::endl;
    for (int64_t k = 0; k < Nz; k++) {
      for (int64_t j = 0; j < Ny; j++) {
        for (int64_t i = 0, l = ((Nx / procs) * rank); i < (Nx / procs), l < ((Nx / procs) * (rank + 1)); i++, l++) {

          double freq = 1.0; /*  */

          double dx     = 1.0 / Nx; // 1 wave divided by number of points
          double x_phys = i * dx;
          double kn     = 2.0 * M_PI * freq;

          data_cpu_in[(k * Ny * (Nz + 2)) + (j * (Nz + 2)) + i].real(
              //ys 8 * ((sin(1.0 * l * dx) * sin(2.0 * j * dy) * sin(3.0 * k * dz)) +
              //ys      (sin(4.0 * l * dx) * sin(5.0 * j * dy) * sin(6.0 * k * dz)));
              8 * sin(kn * x_phys));
          data_cpu_in[(k * Ny * (Nz + 2)) + (j * (Nz + 2)) + i].imag(0.0);
        }
      }
    }
    for (int64_t k = 0; k < Nz; k++) {
      for (int64_t j = 0; j < Ny; j++) {
        for (int64_t i = 0, l = ((Nx / procs) * rank); i < (Nx / procs), l < ((Nx / procs) * (rank + 1)); i++, l++) {

          auto& value = data_cpu_in[(k * Ny * (Nz + 2)) + (j * (Nz + 2)) + i];
          std::cout << "data_cpu_in[" << i << "," << j << "," << k << "] = "
                    << "Real: " << value.real() << ", Imag: " << value.imag() << std::endl;
        }
      }
    }

    std::cout << "---------------------test FFT----------------" << std::endl;
  }

  {
    const int    Nx = 120.0;
    const int    Ny = 2.0;
    const int    Nz = 2.0;
    const double Lx = 1.0;
    //     index = x + Nx*(y + Ny*z)
    hipfftReal* data_cpu = new hipfftReal[Nx * Ny * Nz];
    for (int z = 0; z < Nz; ++z) {
      for (int y = 0; y < Ny; ++y) {
        for (int x = 0; x < Nx; ++x) {
          int idx = x + Nx * (y + Ny * z);

          double freq = 240.0; /*  */

          double dx     = 1.0 / Nx; // 1 wave divided by number of points
          double x_phys = x * dx;
          double k      = 2.0 * M_PI * freq;

          //ys         data_cpu[idx] = 8.0f * cos((2.0 * M_PI *freq) * Lx / Nx * x);

          data_cpu[idx] = 8.0f * cos(k * x_phys);
          std::cout << "data_cpu[" << x << "][" << y << "][" << z << "] = " << data_cpu[idx] << std::endl;
        }
      }
    }

    hipfftReal* data_in_gpu;
    hipMalloc((void**)&data_in_gpu, Nx * Ny * Nz * sizeof(hipfftReal));
    hipMemcpy(data_in_gpu, data_cpu, Nx * Ny * Nz * sizeof(hipfftReal), hipMemcpyHostToDevice);

    hipfftComplex* data_out_gpu;
    hipMalloc((void**)&data_out_gpu, (Nx / 2 + 1) * Ny * Nz * sizeof(hipfftComplex));

    hipfftHandle plan;
    hipfftPlan1d(&plan, Nx, HIPFFT_R2C, Ny * Nz);

    hipfftExecR2C(plan, data_in_gpu, data_out_gpu);

    hipfftComplex* result = new hipfftComplex[(Nx / 2 + 1) * Ny * Nz];
    hipMemcpy(result, data_out_gpu, (Nx / 2 + 1) * Ny * Nz * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for (int by = 0; by < Ny; ++by) {
      for (int bz = 0; bz < Nz; ++bz) {
        int batchIndex = by + Ny * bz;
        std::cout << "=== batch (y=" << by << ", z=" << bz << ") ===" << std::endl;

        for (int k = 0; k < Nx / 2 + 1; ++k) {

          int idx = batchIndex * (Nx / 2 + 1) + k;

          float re = result[idx].x / Nx;
          float im = result[idx].y / Nx;

          std::cout << "  k=" << k << "  Re=" << re << "  Im=" << im << std::endl;
        }
      }
    }

    double E_time = 0.0;
    for (int n = 0; n < Nx * Ny * Nz; ++n) {
      E_time += (data_cpu[n] * data_cpu[n]);
    }

    double E_freq = 0.0;

    for (int by = 0; by < Ny; ++by) {
      for (int bz = 0; bz < Nz; ++bz) {
        int batchIndex = by + Ny * bz;
        for (int k = 0; k < Nx / 2 + 1; ++k) {
          int    idx  = batchIndex * (Nx / 2 + 1) + k;
          double re   = result[idx].x;
          double im   = result[idx].y;
          double mag2 = re * re + im * im;

          E_freq += 2.0 * mag2;
        }
      }
    }

    E_freq /= Nx;

    std::cout << "[Time domain energy] E_time = " << E_time << std::endl;
    std::cout << "[Freq domain energy] E_freq = " << E_freq << std::endl;

    hipfftDestroy(plan);
    hipFree(data_in_gpu);
    hipFree(data_out_gpu);
    delete[] data_cpu;
    delete[] result;

    return 0;
  }
}
